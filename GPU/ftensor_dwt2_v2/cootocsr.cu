#include "cootocsr.h"
void cootocsr(int m,int nnz,int* c,int* rrow){
	hipsparseHandle_t handle;
	hipError_t stat1=hipSuccess;
	hipError_t stat2=hipSuccess;
	hipError_t stat3=hipSuccess;
	hipError_t stat4=hipSuccess;
	hipsparseStatus_t stat6=HIPSPARSE_STATUS_SUCCESS;
	hipsparseStatus_t stat5=HIPSPARSE_STATUS_SUCCESS;
	hipsparseStatus_t stat7=HIPSPARSE_STATUS_SUCCESS;
	int* row;
	int* crow;
	stat1=hipMalloc((void**)&crow,sizeof(int)*nnz);
	stat2=hipMalloc((void**)&row,sizeof(int)*(m+1));
	stat3=hipMemcpy(crow,c,sizeof(int)*nnz,hipMemcpyHostToDevice);
	stat6=hipsparseCreate(&handle);
	stat5=hipsparseXcoo2csr(handle,crow,nnz,m,row,HIPSPARSE_INDEX_BASE_ZERO);
	stat4=hipMemcpy(rrow,row,sizeof(int)*(m+1),hipMemcpyDeviceToHost);
	stat7=hipsparseDestroy(handle);
	if(stat1!=hipSuccess||
			stat2!=hipSuccess||
			stat3!=hipSuccess||
			stat4!=hipSuccess){
		printf("runtime API error\n");
		exit(-1);
	}
	if(stat5!=HIPSPARSE_STATUS_SUCCESS||
			stat6!=HIPSPARSE_STATUS_SUCCESS||
			stat7!=HIPSPARSE_STATUS_SUCCESS){
		printf("cuSparse API error\n");
		exit(-1);
	}
	hipFree(row);
	hipFree(crow);
}
