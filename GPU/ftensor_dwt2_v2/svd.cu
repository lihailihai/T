#include"svd.h"
#include<hipsolver.h>
void svd(int m,int n,hipComplex* T,hipComplex* U,hipComplex* V,float* S){
     hipsolverHandle_t handle;
     hipsolverGesvdjInfo_t params=NULL;
     int* info=NULL;
     int echo=1;
     int lda=0;
     lda=m;
     int ldu=0;
     ldu=m;
     int ldv=0;
     ldv=n;
     int lwork=0;
     hipComplex* work=NULL;
     float* s=NULL;
     hipComplex* u=NULL;
     hipComplex* v=NULL;
     hipComplex* t=NULL;
     hipsolverStatus_t status=HIPSOLVER_STATUS_SUCCESS;
     status=hipsolverDnCreate(&handle);
     assert(status==HIPSOLVER_STATUS_SUCCESS);
     status=hipsolverDnCreateGesvdjInfo(&params);
     assert(status==HIPSOLVER_STATUS_SUCCESS);
     hipError_t stat1=hipSuccess;
     hipError_t stat2=hipSuccess;
     hipError_t stat3=hipSuccess;
     hipError_t stat4=hipSuccess;
     hipError_t stat5=hipSuccess;
     hipError_t stat6=hipSuccess;
     stat1=hipMalloc((void**)&info,sizeof(int));
     int* inf=(int*)malloc(sizeof(int));
     stat2=hipMalloc((void**)&u,sizeof(hipComplex)*m*((m<n)?m:n));
     stat3=hipMalloc((void**)&v,sizeof(hipComplex)*n*((m<n)?m:n));
     stat4=hipMalloc((void**)&s,sizeof(float)*((m<n)?m:n));
     stat5=hipMalloc((void**)&t,sizeof(hipComplex)*m*n);
     stat6=hipMemcpy(t,T,sizeof(hipComplex)*m*n,hipMemcpyHostToDevice);
     if(
    		 stat1!=hipSuccess||
    		 stat2!=hipSuccess||
    		 stat3!=hipSuccess||
    		 stat4!=hipSuccess||
    		 stat5!=hipSuccess||
    		 stat6!=hipSuccess){
    	 printf("cuda malloc error\n");
    	 exit(-1);
     }
     if(hipsolverDnCgesvdj_bufferSize(
    		 handle,
    		 HIPSOLVER_EIG_MODE_VECTOR,
    		 echo,
    		 m,
    		 n,
    		 t,
    		 m,
    		 s,
    		 u,
    		 ldu,
    		 v,
    		 ldv,
    		 &lwork,
    		 params)!=HIPSOLVER_STATUS_SUCCESS){
    	 printf("hipsolverDnCgesvdj_bufferSize failed\n");
    	 exit(-1);

     }
     if(hipDeviceSynchronize()!=hipSuccess){
    	 printf("synchronize failed");
    	 exit(-1);
     }
     stat1=hipMalloc((void**)&work,sizeof(hipComplex)*lwork);
     assert(stat1==hipSuccess);
     if(hipsolverDnCgesvdj(
    		 handle,
    		 HIPSOLVER_EIG_MODE_VECTOR,
    		 echo,
    		 m,
    		 n,
    		 t,
    		 lda,
    		 s,
    		 u,
    		 ldu,
    		 v,
    		 ldv,
    		 work,
    		 lwork,
    		 info,
    		 params)!=HIPSOLVER_STATUS_SUCCESS){
    	 printf("hipsolverDnCgesvdj err\n");
    	 return;
     }
     if(hipDeviceSynchronize()!=hipSuccess){
    	 printf("cuda synchronize err\n");
    	 return;
     }
     stat1=hipMemcpy(U,u,sizeof(hipComplex)*m*((m<n)?m:n),hipMemcpyDeviceToHost);
     assert(stat1==hipSuccess);
     stat1=hipMemcpy(V,v,sizeof(hipComplex)*n*((m<n)?m:n),hipMemcpyDeviceToHost);
     assert(stat1==hipSuccess);
     stat1=hipMemcpy(S,s,sizeof(float)*((m<n)?m:n),hipMemcpyDeviceToHost);
     assert(stat1==hipSuccess);
     hipMemcpy(inf,info,sizeof(int),hipMemcpyDeviceToHost);
     printf("info %d\n",*inf);
     free(inf);
     stat1=hipFree(u);
     assert(stat1==hipSuccess);
     stat1=hipFree(v);
     assert(stat1==hipSuccess);
     stat1=hipFree(s);
     assert(stat1==hipSuccess);
     hipFree(info);
     hipFree(work);
     status=hipsolverDnDestroy(handle);
     assert(status==HIPSOLVER_STATUS_SUCCESS);
     status=hipsolverDnDestroyGesvdjInfo(params);
     assert(status==HIPSOLVER_STATUS_SUCCESS);
     printf("svd success\n");
}

void svd_getv(int m,int n,hipComplex *A,hipComplex *VT,hipComplex* U,float* S){
	hipsolverHandle_t cusolverH = NULL;
	hipblasHandle_t cublasH = NULL;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	hipError_t cudaStat1 = hipSuccess;
	hipError_t cudaStat2 = hipSuccess;
	hipError_t cudaStat3 = hipSuccess;
	hipError_t cudaStat4 = hipSuccess;
	hipError_t cudaStat5 = hipSuccess;



	int lda=m,ldu=m,ldvt=n;
	hipComplex *d_A;
    hipComplex *d_U;
  	float *d_S;
  	hipComplex *d_VT;
  	int *devInfo=NULL;
  	hipComplex *d_work = NULL;
  	float *r_work = NULL;
  	int lwork =0;

  	int info_gpu = 0;
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	cublas_status = hipblasCreate(&cublasH);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	//copy A U S and VT to device
	cudaStat1 = hipMalloc((void**)&d_A, sizeof(hipComplex)*lda*n);
	cudaStat2 = hipMalloc((void**)&d_U, sizeof(hipComplex)*ldu*m);
	cudaStat3 = hipMalloc((void**)&d_S, sizeof(float)*n);
	cudaStat4 = hipMalloc((void**)&d_VT, sizeof(hipComplex)*ldvt*n);
	cudaStat5 = hipMalloc((void**)&devInfo, sizeof(int));

	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);
	assert(hipSuccess == cudaStat4);
	assert(hipSuccess == cudaStat5);

	cudaStat1 = hipMemcpy(d_A, A, sizeof(hipComplex)*lda*n, hipMemcpyHostToDevice);
/*	cudaStat2 = hipMemcpy(d_U, U, sizeof(hipComplex)*ldu*m, hipMemcpyHostToDevice);
	cudaStat3 = hipMemcpy(d_S, S, sizeof(float)*n, hipMemcpyHostToDevice);
	cudaStat4 = hipMemcpy(d_VT, VT, sizeof(hipComplex)*ldvt*n, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);
	assert(hipSuccess == cudaStat4);*/

	cusolver_status = hipsolverDnCgesvd_bufferSize(
		cusolverH,
		m,
		n,
		&lwork);
	assert(hipSuccess == cudaStat1);

	cudaStat1 = hipMalloc((void**)&d_work, sizeof(hipComplex)*lwork);
	cudaStat3 = hipMalloc((void**)&r_work, sizeof(float)*lwork);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat3);

	//compute SVD
	cusolver_status = hipsolverDnCgesvd(
		cusolverH,
		'A',
		'A',
		m,
		n,
		d_A,
		lda,
		d_S,
		d_U,
		ldu,
		d_VT,
		ldvt,
		d_work,
		lwork,
		r_work,
		devInfo
		);

	cudaStat1 = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat1);

	//check if SVD is good or not
	cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat1);
	assert(0 == info_gpu);

	//copy U S and VT to host
	cudaStat1 = hipMemcpy(U, d_U, sizeof(hipComplex)*ldu*m, hipMemcpyDeviceToHost);
	 cudaStat2 = hipMemcpy(S, d_S, sizeof(float)*n, hipMemcpyDeviceToHost);
	cudaStat3 = hipMemcpy(VT, d_VT, sizeof(hipComplex)*ldvt*n, hipMemcpyDeviceToHost);
	 assert(hipSuccess == cudaStat1);
	 assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);

	if (d_A) hipFree(d_A);
	if (devInfo) hipFree(devInfo);
	if (d_work) hipFree(d_work);
	if (r_work) hipFree(r_work);

	if (cublasH) hipblasDestroy(cublasH);
	if (cusolverH) hipsolverDnDestroy(cusolverH);

	hipDeviceReset();
}
