#include "hip/hip_runtime.h"

#include<hip/hip_runtime.h>
#include<malloc.h>
#include"gettime.c"
#include"tensor.c"
#include<assert.h>
#include"svd.h"
#include"check.h"
#include"multi_mm.h"
#include"kernel.h"
#include<stdio.h>
#include"dwt.cuh"

int main(int argc,char* argv[]){

	extern void cootocsr(int,int,int*,int*);
	int m1,n1,k1,l1;
	printf("input the size of tensor(m*n*k*l):");
	scanf("%d%d%d%d",&m1,&n1,&k1,&l1);
	printf("m=%d,n=%d,k=%d,l=%d\n",m1,n1,k1,l1);
	int m=m1;
    int n=n1;
    int k=((k1+7)/2)*2;
	int l=((l1+7)/2)*2;
	int k2=((k1+7)/2);
    int l2=((l1+7)/2);
	/*===========read rgbdata==================*/
	printf("data loading………………………\n");
    FILE* f=NULL;
    const char* F_PATH="/home/haili/Documents/MATLAB/kobe.txt";
    f = fopen(F_PATH,"r");
    if(!f){
    	printf("Error when open file\n");
    	exit(1);
    }
	char line[100];
	int num=0;
	float ten;
	hipComplex* tensor=(hipComplex *)malloc(sizeof(hipComplex)*m*n*k1*l1);
	for(int i=0;i<m*n*k1*l1;i++){
	    if(fgets(line,100,f)!=NULL){
		sscanf(line,"%f",&ten);
		tensor[num].x=ten;
		tensor[num].y=0;
		num++;
	}
		}
    printf("count:%d \n",num);
    fclose(f);
	hipComplex* w=(hipComplex*)malloc(m*n*sizeof(hipComplex));
	//store
	float* S=(float*)malloc(sizeof(float)*((m<n)?m:n));
	hipComplex* U=(hipComplex*)malloc(sizeof(hipComplex)*m*m);
	hipComplex* V=(hipComplex*)malloc(sizeof(hipComplex)*n*n);

	float* S1=(float*)malloc(sizeof(float)*((m<n)?m:n)*k*l);
	hipComplex* U1=(hipComplex*)malloc(sizeof(hipComplex)*k*l*m*((m<n)?m:n));
	hipComplex* V1=(hipComplex*)malloc(sizeof(hipComplex)*k*l*n*((m<n)?m:n));

	hipComplex* U2=(hipComplex*)malloc(sizeof(hipComplex)*k*l*m*((m<n)?m:n));
	hipComplex* V2=(hipComplex*)malloc(sizeof(hipComplex)*k*l*n*((m<n)?m:n));


	hipfftComplex* A=(hipfftComplex*)malloc(m*n*k*l*sizeof(hipfftComplex));
	hipfftComplex* B=(hipfftComplex*)malloc(m*n*k*l*sizeof(hipfftComplex));
	hipfftComplex* C=(hipfftComplex*)malloc(m*n*k*l*sizeof(hipfftComplex));
	hipfftComplex* D=(hipfftComplex*)malloc(m*n*k*l*sizeof(hipfftComplex));
	hipfftComplex* E=(hipfftComplex*)malloc(m*n*k*l*sizeof(hipfftComplex));
	hipfftComplex* F=(hipfftComplex*)malloc(m*n*k*l*sizeof(hipfftComplex));
	hipfftComplex* G=(hipfftComplex*)malloc(m*n*k1*l1*sizeof(hipfftComplex));
	if(!w||!A||!C||!D||!S||!U||!V){
		printf("host memory allocation failed");
		exit(-1);
	}
	double time1,time2,time3;
	time1=gettime();
	printf(" shishu tensor\n");
	//w=createtensor(m,n,k,l,w);
//	printtensor(m,n,k,l,w);
	//free(w);
	//create  4D tensor of size m*n*k*l
	/*A=createctensor(m,n,k,l,A);
	printf("cucomplex tensor\n");
    printctensor(m,n,k,l,A);*/
    //transform tensor to tensor_scalar
	int threads;
	int blocks;
	hipComplex* A1;
	hipComplex* A2;
	hipComplex* A6;
	hipComplex* A7;
	float* B1;
	float* B2;
	float* A3=(float*)malloc(sizeof(float)*m*n*k1*l1);
	float* A4=(float*)malloc(sizeof(float)*m*n*k*l);
	float* A5=(float*)malloc(sizeof(float)*m*n*k*l);
	hipMalloc((void**)&B1,sizeof(float)*n*m*k*l);
	hipMalloc((void**)&B2,sizeof(float)*n*m*k1*l1);
	hipMalloc((void**)&A1,sizeof(hipComplex)*n*m*k1*l1);
	hipMalloc((void**)&A2,sizeof(hipComplex)*n*m*k1*l1);
	hipMalloc((void**)&A6,sizeof(hipComplex)*n*m*k*l);
	hipMalloc((void**)&A7,sizeof(hipComplex)*n*m*k*l);
	hipMemcpy(A1,tensor,sizeof(hipComplex)*m*n*k1*l1,hipMemcpyHostToDevice);
    if(m*n*k1*l1<512){
    	threads=m*k1*n*l1;
    	blocks=1;
    }else{
    	threads=512;
    	blocks=(m*n*k1*l1%512==0)?m*n*k1*l1/512:m*n*k1*l1/512+1;
    }
    d_tensor_scalar<<<blocks,threads>>>(A2,A1,m,n,k1,l1);
    hipMemcpy(G,A2,sizeof(hipComplex)*m*n*k1*l1,hipMemcpyDeviceToHost);
	printf("out put G\n");
	for(int i=0;i<m*n*k1*l1;i++){
			A3[i]=G[i].x;
			}
	//printctensor(m,n,k,l,G);
	//hipFree(A1);hipFree(A2);

   // C=tensor_scalar(m,n,k,l,tensor,C);


   /* for(int i=0;i<m*n*k*l;i++){
    printf("%f%s%fi\n",C[i].x-G[i].x,
    		(C[i].y-G[i].y)<0?"":"+",C[i].y-G[i].y);
    }*/
   
   // hipMemcpy(G,A1,sizeof(hipComplex)*m*n*k*l,hipMemcpyDeviceToHost);
    printf("test tensor_scalar to tensor\n");
   /* for(int i=0;i<m*n*k*l;i++){
        printf("%f%s%fi\n",A[i].x-G[i].x,
        		(A[i].y-G[i].y)<0?"":"+",A[i].y-G[i].y);
        }*/
    printf("tensor scalar \n");
  //  printctensor(m,n,k,l,C);
    //take  2Ddwt
    int block=l1*m*n;
    float* swap3;
    float* swap1;
    float * E1=(float*)malloc(sizeof(float)*m*n*k*l1);
    float * E2=(float*)malloc(sizeof(float)*m*n*k*l1);
    hipMalloc((void**)&swap3,sizeof(float)*m*n*k*l1);
    hipMalloc((void**)&swap1,sizeof(float)*m*n*k1*l1);
    hipMemcpy(swap1,A3,sizeof(float)*n*m*k1*l1,hipMemcpyHostToDevice);
    dwt_batch<<<block,THREADS>>>(swap1,k1,l1*m*n,swap3);
    hipMemcpy(E1,swap3,sizeof(float)*m*n*k*l1,hipMemcpyDeviceToHost);
    t_scalar_transpose(m,n,k,l1,E1,E2);
   /* for(int i=0;i<m*n*k*l1;i++){
       printf("%f\n",E2[i]);}*/
    block=k*m*n;
    float* swap6;
    hipMalloc((void**)&swap6,sizeof(float)*m*n*k*l);
    hipMemcpy(swap3,E2,sizeof(float)*m*n*k*l1,hipMemcpyHostToDevice);
    dwt_batch<<<block,THREADS>>>(swap3,l1,k*n*m,swap6);
    hipMemcpy(A4,swap6,sizeof(float)*m*n*k*l,hipMemcpyDeviceToHost);
    t_scalar_transpose(m,n,l,k,A4,A5);
    for(int d=0;d<k*l*m*n;d++){
        		B[d].x=A5[d];
        		B[d].y=0;
        	}
  //  fft_batch(m,n,k,l,G,B);
    printf("output batch dwt…………………………………………………………………………\n");
  //  printctensor(m,n,k,l,B);
    //transform tensor_scalar to tensor
    //E=tensor_scalartotensor(m,n,k,l,B,E);
    if(m*n*k*l<512){
        	threads=m*k1*n*l1;
        	blocks=1;
        }else{
        	threads=512;
        	blocks=(m*n*k*l%512==0)?m*n*k*l/512:m*n*k*l/512+1;
        }
    hipMemcpy(A6,B,sizeof(hipComplex)*m*n*k*l,hipMemcpyHostToDevice);
    d_tensor_scalar_tensor<<<blocks,threads>>>(A7,A6,m,n,k,l);
    printf("tensor scalar to tensor\n");
   // printctensor(m,n,k,l,E);
   // matview_transpose(m,n,k,l,E,F);
   // hipMemcpy(A2,E,sizeof(hipComplex)*m*n*k*l,hipMemcpyHostToDevice);
    d_matview_transpose<<<blocks,threads>>>(A6,A7,m,n,k,l);
    hipMemcpy(F,A6,sizeof(hipComplex)*m*n*k*l,hipMemcpyDeviceToHost);
    printf("test matviw_transpose………………………………………………\n");
 /*   for(int i=0;i<m*n*k*l;i++){
               printf("%f%s%fi\n",F[i].x-G[i].x,
               		(F[i].y-G[i].y<0)?"":"+",F[i].y-G[i].y);
               }
        for(int i=0;i<m*n*k*l;i++){
            printf("%f%s%fi\n",G[i].x,
            		G[i].y<0?"":"+",G[i].y);
            }*/
    printf("……………………………………………………………………………………………………\n");
   // printctensor(m,n,k,l,F);
    //take SVD
    hipfftComplex** pmm=(hipfftComplex**)F;
    hipfftComplex (*pm)[m*n]=(hipfftComplex (*)[m*n])pmm;
    hipfftComplex* temp=(hipfftComplex*)malloc(sizeof(hipfftComplex)*m*n);
    for(int q=0;q<k*l;q++){
    for(int i=0;i<m*n;i++){
    	temp[i]=pm[q][i];
    }
   /* for(int i=0;i<m*n;i++){
       	printf("temp %f %f\n",temp[i].x,temp[i].y);
       }*/
    svd(m,n,(hipComplex*)temp,U,V,S);

    for(int j=0;j<m*((m<n)?m:n);j++){

    	 U2[j+q*m*((m<n)?m:n)].x=U[j].x;
    	 U2[j+q*m*((m<n)?m:n)].y=U[j].y;

     }

    for(int j=0;j<n*((m<n)?m:n);j++){

        V2[j+q*n*((m<n)?m:n)].x=V[j].x;
        V2[j+q*n*((m<n)?m:n)].y=V[j].y;
    }

   for(int j=0;j<((m<n)?m:n);j++){
    	S1[j+q*((m<n)?m:n)]=S[j];
    }

    }
    //output S1 U1 V1
  //  printSVD(m,n,k,l,S1,U2,V2);
    matview_transpose(((m<n)?m:n),m,k,l,U2,U1);
    //U1 to coo
    int count=0;
    for(int i=0;i<m*((m<n)?m:n)*k*l;i++){
    if(U1[i].x!=0||U1[i].y!=0){
     count++;
    }}
    printf("output U1 count %d\n",count);
    int* row_array=(int*)malloc(sizeof(int)*count);
    int* col_array=(int*)malloc(sizeof(int)*count);
    hipComplex* data_array=(hipComplex*)malloc(sizeof(hipComplex)*count);
    coo* t1coo=(coo*)malloc(sizeof(coo));
    coo* tcoo=(coo*)malloc(sizeof(coo));
    tcoo=matviewtocoo(m,((m<n)?m:n),k,l,t1coo,U1,row_array,col_array,data_array);
   /* for(int i=0;i<count;i++){
    	printf("%d %d %f%s%fi\n",tcoo->row_array[i],
    			tcoo->col_array[i],tcoo->data_array[i].x,
    			((tcoo->data_array[i].y<0)?"":"+"),tcoo->data_array[i].y);
    }*/


    //coo to csr
    csr* tcsr_U=(csr*)malloc(sizeof(csr));
    int* rrow=(int*)malloc(sizeof(int)*(m*k*l+1));
   /* for(int i=0;i<count;i++){
    printf("%d\n",tcoo->row_array[i]);}*/
    cootocsr((m*k*l),count,tcoo->row_array,rrow);
    tcsr_U->col_array=NULL;
    tcsr_U->data_array=NULL;
    tcsr_U->row_array=rrow;
    printf("output  U2 csr->row_array\n");
    /*for(int i=0;i<(m*k*l+1);i++){
   	printf("%d\n",tcsr_U->row_array[i]);
    }*/



    count=0;
    //V1 to coo
  /*  matview_transpose(((m<n)?m:n),n,k,l,V2,V1);
    free(V2);
    for(int i=0;i<n*((m<n)?m:n)*k*l;i++){
        if(V1[i].x!=0||V1[i].y!=0){
         count++;
        }}*/
    for(int i=0;i<n*((m<n)?m:n)*k*l;i++){
            if(V2[i].x!=0||V2[i].y!=0){
             count++;}}
    printf("output V1 count %d\n",count);
    int* row_array1=(int*)malloc(sizeof(int)*count);
    int* col_array1=(int*)malloc(sizeof(int)*count);
    hipComplex* data_array1=(hipComplex*)malloc(sizeof(hipComplex)*count);
    coo* tcoo2=(coo*)malloc(sizeof(coo));
    coo* tcoo3=(coo*)malloc(sizeof(coo));
    tcoo3=matviewtocoo(((m<n)?m:n),n,k,l,tcoo3,V2,row_array1,col_array1,data_array1);
   /* for(int i=0;i<count;i++){
        	printf("%d %d %f%s%fi\n",tcoo3->row_array[i],
        			tcoo3->col_array[i],tcoo3->data_array[i].x,
        			((tcoo3->data_array[i].y<0)?"":"+"),tcoo3->data_array[i].y);
        }*/
//V coo to csr
    csr* tcsr_V=(csr*)malloc(sizeof(csr));
    int* rrow1=(int*)malloc(sizeof(int)*(((m<n)?m:n)*k*l+1));
       cootocsr(((m<n)?m:n)*k*l,count,tcoo3->row_array,rrow1);
       tcsr_V->row_array=rrow1;
       printf("output  V2 csr->row_array\n");
     /*  for(int i=0;i<(((m<n)?m:n)*k*l+1);i++){
       	printf("%d\n",tcsr_V->row_array[i]);
       }*/


    count=0;
    //S1 to coo

    for(int i=0;i<((m<n)?m:n)*k*l;i++){
                if(S1[i]!=0){
                 count++;
               //  printf("qian %f\n",S1[i]);
                }
        }
    /*====================================================================set r=====================================================*/
    printf("output S1 count %d\n",count);
 //   cdata* pg;
   // int r=0;
    int r=((m<n)?m:n)*k*l*9/10;
//    float* d_S1=NULL;
    cdata* pg1=(cdata*)malloc(sizeof(cdata)*((m<n)?m:n)*k*l);
    for(int i=0;i<((m<n)?m:n)*k*l;i++){
        pg1[i].a=S1[i];
        pg1[i].b=i;
      }
      cdata swap;
     for(int i=0;i<((m<n)?m:n)*k*l;i++){
        for(int j=i+1;j<((m<n)?m:n)*k*l;j++){
        if(pg1[i].a>pg1[j].a){
         swap.a=pg1[i].a;
         swap.b=pg1[i].b;
         pg1[i].a=pg1[j].a;
         pg1[i].b=pg1[j].b;
         pg1[j].a=swap.a;
         pg1[j].b=swap.b;
       }
      } 
     }
     for(int i=0;i<r;i++){
      S1[pg1[i].b]=0;
     }
   /* for(int i=0;i<((m<n)?m:n)*k*l;i++){
        pg1[i].a=S1[i];
        pg1[i].b=i;
      }
    hipMalloc((void**)&d_S1,sizeof(float)*((m<n)?m:n)*k*l);
    hipMalloc((void**)&pg,sizeof(cdata)*((m<n)?m:n)*k*l);
    hipMemcpy(d_S1,S1,sizeof(float)*((m<n)?m:n)*k*l,hipMemcpyHostToDevice);
    hipMemcpy(pg,pg1,sizeof(float)*((m<n)?m:n)*k*l,hipMemcpyHostToDevice);
    if(m*n*k*l<512){
        	threads=m*k*n*l;
        	blocks=1;
        }else{
        	threads=512;
        	blocks=((((m<n)?m:n)*k*l)%threads==0)?((m<n)?m:n)*k*l/threads:(((m<n)?m:n)*k*l/threads+1);
        }
    odd_even<<<blocks,threads>>>(d_S1,pg,((m<n)?m:n)*k*l,r);
    hipMemcpy(S1,d_S1,sizeof(float)*((m<n)?m:n)*k*l,hipMemcpyDeviceToHost);
    hipMemcpy(pg1,pg,sizeof(cdata)*((m<n)?m:n)*k*l,hipMemcpyDeviceToHost);*/
    
    count=0;
    for(int i=0;i<((m<n)?m:n)*k*l;i++){
            if(S1[i]!=0){
             count++;

            }
           // printf("hou %f   %f,%d\n",S1[i],pg1[i].a,pg1[i].b);
    }

    printf("output S1 count %d\n",count);
    int* row_array2=(int*)malloc(sizeof(int)*count);
    int* col_array2=(int*)malloc(sizeof(int)*count);
    hipComplex* data_array2=(hipComplex*)malloc(sizeof(hipComplex)*count);
    coo* tcoo4=(coo*)malloc(sizeof(coo));
    for(int i=0,j=0;i<((m<n)?m:n)*k*l;i++){
    	if(S1[i]!=0){
    	row_array2[j]=i;
    	col_array2[j]=i;
    	data_array2[j].x=S1[i];
    	data_array2[j].y=0.000000;
    	j++;
    	}
    }
    tcoo4->row_array=row_array2;
    tcoo4->col_array=col_array2;
    tcoo4->data_array=data_array2;
   /* for(int i=0;i<count;i++){
            	printf("%d %d %f\n",tcoo4->row_array[i],
            			tcoo4->col_array[i],tcoo4->data_array[i].x);
            }*/


    csr* tcsr_S=(csr *)malloc(sizeof(csr));
    tcsr_S->row_array=(int *)malloc(sizeof(int)*(((m<n)?m:n)*k*l+1));
    cootocsr(((m<n)?m:n)*k*l,count,tcoo4->row_array,tcsr_S->row_array);
    count=0;
    /*for(int i=0;i<(((m<n)?m:n)*k*l+1);i++){

    	tcsr_S->row_array[i]=i;
    }*/

  /*  for(int i=0;i<(((m<n)?m:n)*k*l+1);i++){
	   printf("%d\n",tcsr_S->row_array[i]);}*/
    //multi-mm
    //U * S
    hipsparseOperation_t transA=HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t transB=HIPSPARSE_OPERATION_NON_TRANSPOSE;
    int min=((m<n)?m:n);
    int nnzA=0,nnzB=0,nnzC=0,baseC=0;
    	int* C_row_array;
    	int* C_col_array;
    	hipComplex* C_data_array;
    	hipComplex* d_A_data_array;
    	hipComplex* d_B_data_array;
    	hipComplex* d_C_data_array;
    	int* d_A_row_array;
    	int* d_A_col_array;
    	int* d_B_row_array;
    	int* d_B_col_array;
    	int* d_C_row_array;
    	int* d_C_col_array;
        hipError_t stat1=hipSuccess;
        hipError_t stat2=hipSuccess;
        hipError_t stat3=hipSuccess;
        hipError_t stat4=hipSuccess;
        hipError_t stat5=hipSuccess;
        hipError_t stat6=hipSuccess;
        nnzA=tcsr_U->row_array[m*k*l]-tcsr_U->row_array[0];
        nnzB=tcsr_S->row_array[min*k*l]-tcsr_S->row_array[0];

    	stat1=hipMalloc((void**)&d_A_row_array,sizeof(int)*(m*k*l+1));

    	stat2=hipMalloc((void**)&d_A_col_array,sizeof(int)*nnzA);

    	stat3=hipMalloc((void**)&d_A_data_array,sizeof(hipComplex)*nnzA);

    	stat4=hipMalloc((void**)&d_B_row_array,sizeof(int)*(min*k*l+1));
    	stat5=hipMalloc((void**)&d_B_col_array,sizeof(int)*nnzB);
    	stat6=hipMalloc((void**)&d_B_data_array,sizeof(hipComplex)*nnzB);
    	if(
    			stat1!=hipSuccess||
    			stat2!=hipSuccess||
    			stat3!=hipSuccess||
    			stat4!=hipSuccess||
    			stat5!=hipSuccess||
    			stat6!=hipSuccess){
    		printf("cuda malloc faild\n");
    		return 0;
    	}

    	if(hipMemcpy(
    			d_A_row_array,
    			tcsr_U->row_array,
    			sizeof(int)*(m*l*k+1),
    			hipMemcpyHostToDevice)!=hipSuccess){
    		printf("cuda memcpy err 1\n");
    		exit(-1);
    	}
    	if(hipMemcpy(
    			d_A_col_array,
    			tcoo->col_array,
    			sizeof(int)*nnzA,
    			hipMemcpyHostToDevice)!=hipSuccess){
    		printf("cuda memcpy err 2\n");
    		exit(-1);
    	}
    	if(hipMemcpy(
    			d_A_data_array,
    			tcoo->data_array,
    			sizeof(hipComplex)*nnzA,
    			hipMemcpyHostToDevice)!=hipSuccess){
    		printf("cuda memcpy err 3\n");
    		exit(-1);
    	}
        if(hipMemcpy(
        		d_B_row_array,
        		tcsr_S->row_array,
        		sizeof(int)*(min*k*l+1),
        		hipMemcpyHostToDevice)!=hipSuccess){
        	printf("cuda memcpy err 4\n");
        	exit(-1);
        }
        if(hipMemcpy(
        		d_B_col_array,
        		tcoo4->col_array,
        		sizeof(int)*nnzB,
        		hipMemcpyHostToDevice)!=hipSuccess){
        	printf("cuda memcpy err 5\n");
        	exit(-1);
        }
        if(hipMemcpy(
        		d_B_data_array,
        		tcoo4->data_array,
        		sizeof(hipComplex)*nnzB,
        		hipMemcpyHostToDevice)!=hipSuccess){
        	printf("cuda memcpy err 6\n");
        	exit(-1);
        }

        hipsparseHandle_t handle;
        if(hipsparseCreate(&handle)!=HIPSPARSE_STATUS_SUCCESS){
        	printf("cuaparsecreate handle failed\n");
        	return 0;
        }
        hipsparseMatDescr_t descrA;
        hipsparseMatDescr_t descrB;
        hipsparseMatDescr_t descrC;
        hipsparseStatus_t status=HIPSPARSE_STATUS_SUCCESS;
        status=hipsparseCreateMatDescr(&descrA);
        assert(status==HIPSPARSE_STATUS_SUCCESS);
        status=hipsparseCreateMatDescr(&descrB);
        assert(status==HIPSPARSE_STATUS_SUCCESS);
        status=hipsparseCreateMatDescr(&descrC);
        assert(status==HIPSPARSE_STATUS_SUCCESS);
        status=hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL);
        assert(status==HIPSPARSE_STATUS_SUCCESS);
        status=hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL);
        assert(status==HIPSPARSE_STATUS_SUCCESS);
        status=hipsparseSetMatType(descrC,HIPSPARSE_MATRIX_TYPE_GENERAL);
        assert(status==HIPSPARSE_STATUS_SUCCESS);
        int* nnzTotalDevHostPtr=&nnzC;
        if(hipMalloc((void**)&d_C_row_array,sizeof(int)*(m*k*l+1))!=hipSuccess){
        	printf("cuda malloc error\n");
        	return 0;
        }
        if(hipsparseXcsrgemmNnz(
        		handle,
        		transA,
        		transB,
        		m*k*l,
        		min*k*l,
        		min*k*l,
        		descrA,
        		nnzA,
        		d_A_row_array,
        		d_A_col_array,
        		descrB,
        		nnzB,
        		d_B_row_array,
        		d_B_col_array,
        		descrC,
        		d_C_row_array,
        		nnzTotalDevHostPtr
        		)!=HIPSPARSE_STATUS_SUCCESS){
        	printf("gemmnz error\n");
        	exit(-1);
        }
        if(hipDeviceSynchronize()!=hipSuccess){
        	printf("synchronize error\n");
        	return 0;
        }
        if(NULL!=nnzTotalDevHostPtr){
        	nnzC=*nnzTotalDevHostPtr;
        }
        else{
        	hipMemcpy(
        			&nnzC,
        			d_C_row_array+m*k*l,
        			sizeof(int),
        			hipMemcpyDeviceToHost);
        	hipMemcpy(
        			&baseC,
        			d_C_row_array,
        			sizeof(int),
        			hipMemcpyDeviceToHost);
        	nnzC=-baseC;
        }
        printf("nnzC %d\n",nnzC);
        C_row_array=(int*)malloc(sizeof(int)*(m*k*l+1));
        C_col_array=(int*)malloc(sizeof(int)*nnzC);
        C_data_array=(hipComplex*)malloc(sizeof(hipComplex)*nnzC);
        if(
        		!C_row_array||
        		!C_col_array||
        		!C_data_array){
        	printf("multi_mm malloc error");
        }
        hipError_t status2=hipSuccess;
        status2=hipMalloc((void**)&d_C_col_array,sizeof(int)*nnzC);
        assert(status2==hipSuccess);
        status2=hipMalloc((void**)&d_C_data_array,sizeof(hipComplex)*nnzC);
        assert(status2==hipSuccess);
        if(hipsparseCcsrgemm(
        		handle,
        		transA,
        		transB,
        		m*k*l,
        		min*k*l,
        		min*k*l,
        		descrA,
        		nnzA,
        		d_A_data_array,
        		d_A_row_array,
        		d_A_col_array,
        		descrB,
        		nnzB,
        		d_B_data_array,
        		d_B_row_array,
        		d_B_col_array,
        		descrC,
        		d_C_data_array,
        		d_C_row_array,
        		d_C_col_array
        		)!=HIPSPARSE_STATUS_SUCCESS){
        	printf("csrgemm error\n");
        	exit(-1);
        }
        status2=hipDeviceSynchronize();
        assert(status2==hipSuccess);
        status2=hipMemcpy(
        		C_row_array,
        		d_C_row_array,
        		sizeof(int)*(m*k*l+1),
        		hipMemcpyDeviceToHost);
        assert(status2==hipSuccess);
        status2=hipMemcpy(
        		C_col_array,
        		d_C_col_array,
        		sizeof(int)*nnzC,
        		hipMemcpyDeviceToHost);
        assert(status2==hipSuccess);
        status2=hipMemcpy(
        		C_data_array,
        		d_C_data_array,
        		sizeof(hipComplex)*nnzC,
        		hipMemcpyDeviceToHost);
        assert(status2==hipSuccess);
        status=hipsparseDestroyMatDescr(descrA);
        assert(status==HIPSPARSE_STATUS_SUCCESS);
        status=hipsparseDestroyMatDescr(descrB);
        assert(status==HIPSPARSE_STATUS_SUCCESS);
        status=hipsparseDestroyMatDescr(descrC);
        assert(status==HIPSPARSE_STATUS_SUCCESS);
        status=hipsparseDestroy(handle);
        assert(status==HIPSPARSE_STATUS_SUCCESS);
        status2=hipFree(d_A_row_array);
        assert(status2==hipSuccess);
        status2=hipFree(d_A_col_array);
        assert(status2==hipSuccess);
        status2=hipFree(d_A_data_array);
        assert(status2==hipSuccess);
        status2=hipFree(d_B_row_array);
        assert(status2==hipSuccess);
        status2=hipFree(d_B_col_array);
        assert(status2==hipSuccess);
        status2=hipFree(d_B_data_array);
        assert(status2==hipSuccess);
        hipFree(d_C_col_array);
        hipFree(d_C_row_array);
        hipFree(d_C_data_array);


       /* printf("output C_row_array\n");
          for(int i=0;i<(m*k*l+1);i++){
       	   printf("%d\n",C_row_array[i]);

          }
          printf("nnzc%d\n",nnzC);
          for(int i=0;i<nnzC;i++){
       	   printf("%d %f%s%fi\n",C_col_array[i],C_data_array[i].x,
       			  ((C_data_array[i].y<0)?"":"+"),C_data_array[i].y);

          }*/

   /*multi_mm(tcsr_U->row_array,tcoo->col_array,tcoo->data_array,
    		tcsr_S->row_array,
    		tcoo4->col_array,
    		tcoo4->data_array,
    		m,((m<n)?m:n),((m<n)?m:n),k,l,
    		HIPSPARSE_OPERATION_NON_TRANSPOSE,
    		HIPSPARSE_OPERATION_NON_TRANSPOSE
    		);*/
   // *** * V
          hipsparseOperation_t transA1=HIPSPARSE_OPERATION_NON_TRANSPOSE;
          hipsparseOperation_t transB1=HIPSPARSE_OPERATION_NON_TRANSPOSE;
                nnzB=tcsr_V->row_array[((m<n)?m:n)*k*l]-tcsr_V->row_array[0];
                hipComplex* d_A_data_array1;
                    	hipComplex* d_B_data_array1;
                    	hipComplex* d_C_data_array1;
                    	int* d_A_row_array1;
                    	int* d_A_col_array1;
                    	int* d_B_row_array1;
                    	int* d_B_col_array1;
                    	int* d_C_row_array1;
                    	int* d_C_col_array1;
            	stat1=hipMalloc((void**)&d_A_row_array1,sizeof(int)*(m*k*l+1));
            	printf("lihai");
            	stat2=hipMalloc((void**)&d_A_col_array1,sizeof(int)*nnzC);

            	stat3=hipMalloc((void**)&d_A_data_array1,sizeof(hipComplex)*nnzC);

            	stat4=hipMalloc((void**)&d_B_row_array1,sizeof(int)*(((m<n)?m:n)*k*l+1));
            	stat5=hipMalloc((void**)&d_B_col_array1,sizeof(int)*nnzB);
            	stat6=hipMalloc((void**)&d_B_data_array1,sizeof(hipComplex)*nnzB);
            	if(
            			stat1!=hipSuccess||
            			stat2!=hipSuccess||
            			stat3!=hipSuccess||
            			stat4!=hipSuccess||
            			stat5!=hipSuccess||
            			stat6!=hipSuccess){
            		printf("cuda malloc faild\n");
            		return 0;
            	}

            	if(hipMemcpy(
            			d_A_row_array1,
            			C_row_array,
            			sizeof(int)*(m*l*k+1),
            			hipMemcpyHostToDevice)!=hipSuccess){
            		printf("cuda memcpy err 1\n");
            		exit(-1);
            	}
            	if(hipMemcpy(
            			d_A_col_array1,
            			C_col_array,
            			sizeof(int)*nnzC,
            			hipMemcpyHostToDevice)!=hipSuccess){
            		printf("cuda memcpy err 2\n");
            		exit(-1);
            	}
            	if(hipMemcpy(
            			d_A_data_array1,
            			C_data_array,
            			sizeof(hipComplex)*nnzC,
            			hipMemcpyHostToDevice)!=hipSuccess){
            		printf("cuda memcpy err 3\n");
            		exit(-1);
            	}
                if(hipMemcpy(
                		d_B_row_array1,
                		tcsr_V->row_array,
                		sizeof(int)*(((m<n)?m:n)*k*l+1),
                		hipMemcpyHostToDevice)!=hipSuccess){
                	printf("cuda memcpy err 4\n");
                	exit(-1);
                }
                if(hipMemcpy(
                		d_B_col_array1,
                		tcoo3->col_array,
                		sizeof(int)*nnzB,
                		hipMemcpyHostToDevice)!=hipSuccess){
                	printf("cuda memcpy err 5\n");
                	exit(-1);
                }
                if(hipMemcpy(
                		d_B_data_array1,
                		tcoo3->data_array,
                		sizeof(hipComplex)*nnzB,
                		hipMemcpyHostToDevice)!=hipSuccess){
                	printf("cuda memcpy err 6\n");
                	exit(-1);
                }
                hipsparseHandle_t handle1;
                if(hipsparseCreate(&handle1)!=HIPSPARSE_STATUS_SUCCESS){
                	printf("cuaparsecreate handle failed\n");
                	return 0;
                }

                hipsparseMatDescr_t descrA1;
                hipsparseMatDescr_t descrB1;
                hipsparseMatDescr_t descrC1;
                status=hipsparseCreateMatDescr(&descrA1);
                assert(status==HIPSPARSE_STATUS_SUCCESS);
                status=hipsparseCreateMatDescr(&descrB1);
                assert(status==HIPSPARSE_STATUS_SUCCESS);
                status=hipsparseCreateMatDescr(&descrC1);
                assert(status==HIPSPARSE_STATUS_SUCCESS);
                status=hipsparseSetMatType(descrA1,HIPSPARSE_MATRIX_TYPE_GENERAL);
                assert(status==HIPSPARSE_STATUS_SUCCESS);
                status=hipsparseSetMatType(descrB1,HIPSPARSE_MATRIX_TYPE_GENERAL);
                assert(status==HIPSPARSE_STATUS_SUCCESS);
                status=hipsparseSetMatType(descrC1,HIPSPARSE_MATRIX_TYPE_GENERAL);
                assert(status==HIPSPARSE_STATUS_SUCCESS);
                int nnzD=0;int baseD=0;
                int* nnzTotalDevHostPtr1=&nnzD;
                if(hipMalloc((void**)&d_C_row_array1,sizeof(int)*(m*k*l+1))!=hipSuccess){
                	printf("cuda malloc error\n");
                	return 0;
                }
                if(hipsparseXcsrgemmNnz(
                		handle1,
                		transA1,
                		transB1,
                		m*k*l,
                		n*k*l,
                		min*k*l,
                		descrA1,
                		nnzC,
                		d_A_row_array1,
                		d_A_col_array1,
                		descrB1,
                		nnzB,
                		d_B_row_array1,
                		d_B_col_array1,
                		descrC1,
                		d_C_row_array1,
                		nnzTotalDevHostPtr1
                		)!=HIPSPARSE_STATUS_SUCCESS){
                	printf("gemmnz error\n");
                	exit(-1);
                }
                if(hipDeviceSynchronize()!=hipSuccess){
                	printf("synchronize error\n");
                	return 0;
                }
                if(NULL!=nnzTotalDevHostPtr){
                	nnzD=*nnzTotalDevHostPtr1;
                }
                else{
                	hipMemcpy(
                			&nnzD,
                			d_C_row_array1+m*k*l,
                			sizeof(int),
                			hipMemcpyDeviceToHost);
                	hipMemcpy(
                			&baseD,
                			d_C_row_array1,
                			sizeof(int),
                			hipMemcpyDeviceToHost);
                	nnzD=-baseD;
                }
                printf("nnzD %d\n",nnzD);
              int*  D_row_array=(int*)malloc(sizeof(int)*(m*k*l+1));
              int * D_col_array=(int*)malloc(sizeof(int)*nnzD);
              hipComplex*  D_data_array=(hipComplex*)malloc(sizeof(hipComplex)*nnzD);
                if(
                		!D_row_array||
                		!D_col_array||
                		!D_data_array){
                	printf("multi_mm malloc error");
                }

                status2=hipMalloc((void**)&d_C_col_array1,sizeof(int)*nnzD);
                assert(status2==hipSuccess);
                status2=hipMalloc((void**)&d_C_data_array1,sizeof(hipComplex)*nnzD);
                assert(status2==hipSuccess);
                if(hipsparseCcsrgemm(
                		handle1,
                		transA1,
                		transB1,
                		m*k*l,
                		n*k*l,
                		min*k*l,
                		descrA1,
                		nnzC,
                		d_A_data_array1,
                		d_A_row_array1,
                		d_A_col_array1,
                		descrB1,
                		nnzB,
                		d_B_data_array1,
                		d_B_row_array1,
                		d_B_col_array1,
                		descrC1,
                		d_C_data_array1,
                		d_C_row_array1,
                		d_C_col_array1
                		)!=HIPSPARSE_STATUS_SUCCESS){
                	printf("csrgemm error\n");
                	exit(-1);
                }
                status2=hipDeviceSynchronize();
                assert(status2==hipSuccess);
                status2=hipMemcpy(
                		D_row_array,
                		d_C_row_array1,
                		sizeof(int)*(m*k*l+1),
                		hipMemcpyDeviceToHost);
                assert(status2==hipSuccess);
                status2=hipMemcpy(
                		D_col_array,
                		d_C_col_array1,
                		sizeof(int)*nnzD,
                		hipMemcpyDeviceToHost);
                assert(status2==hipSuccess);
                status2=hipMemcpy(
                		D_data_array,
                		d_C_data_array1,
                		sizeof(hipComplex)*nnzD,
                		hipMemcpyDeviceToHost);
                assert(status2==hipSuccess);
                status=hipsparseDestroyMatDescr(descrA1);
                assert(status==HIPSPARSE_STATUS_SUCCESS);
                status=hipsparseDestroyMatDescr(descrB1);
                assert(status==HIPSPARSE_STATUS_SUCCESS);
                status=hipsparseDestroyMatDescr(descrC1);
                assert(status==HIPSPARSE_STATUS_SUCCESS);
                status=hipsparseDestroy(handle1);
                assert(status==HIPSPARSE_STATUS_SUCCESS);
                status2=hipFree(d_A_row_array1);
                assert(status2==hipSuccess);
                status2=hipFree(d_A_col_array1);
                assert(status2==hipSuccess);
                status2=hipFree(d_A_data_array1);
                assert(status2==hipSuccess);
                status2=hipFree(d_B_row_array1);
                assert(status2==hipSuccess);
                status2=hipFree(d_B_col_array1);
                assert(status2==hipSuccess);
                status2=hipFree(d_B_data_array1);
                assert(status2==hipSuccess);
                if(m*k*l*n<512){
                	threads=m*k*l*n;
                	blocks=1;
                }else{
                	threads=512;
                	blocks=((m*k*l*n)%threads==0)?(m*k*l*n/threads):(m*k*n*l/threads+1);
                }
                d_zero<<<blocks,threads>>>(A6,m*n*k*l);
                if(m*k*l<512){
                                	threads=m*k*l;
                                	blocks=1;
                                }else{
                                	threads=512;
                                	blocks=((m*k*l)%threads==0)?(m*k*l/threads):(m*k*l/threads+1);
                                }
                d_csrtomatview<<<blocks,threads>>>(A6,d_C_data_array1,d_C_row_array1,d_C_col_array1,m,n,k,l);
                hipMemcpy(B,A6,sizeof(hipComplex)*m*n*k*l,hipMemcpyDeviceToHost);
                printf("test csr to matview()\n");
               // printctensor(m,n,k,l,G);
                hipFree(d_C_col_array1);
                hipFree(d_C_row_array1);
                hipFree(d_C_data_array1);
                printf("output D_row_array\n");
                        /* for(int i=0;i<(m*k*l+1);i++){
                      	   printf("%d\n",D_row_array[i]);

                         }*/
                         printf("nnzD%d\n",nnzD);
                        /* for(int i=0;i<nnzD;i++){
                      	   printf("%d %f%s%fi\n",D_col_array[i],(D_data_array[i].x-G[i].x),
                      			  (((D_data_array[i].y)-G[i].y<0)?"":"+"),
		                        (D_data_array[i].y-G[i].y));

                         }*/
    //take iDWT2
    C=tensor_scalar(m,n,k,l,B,C);
   // printctensor(m,n,k,l,C);
    for(int i=0;i<m*n*k*l;i++){
    			A5[i]=C[i].x;
    			}
    t_scalar_transpose(m,n,k,l,A5,A4);
       block=k*m*n;
	   hipMemcpy(swap6,A4,sizeof(float)*m*n*k*l,hipMemcpyHostToDevice);
	   idwt_batch<<<block,THREADS>>>(swap6,l2,l1,k*m*n,swap3);
	   hipMemcpy(E1,swap3,sizeof(float)*m*n*k*l1,hipMemcpyDeviceToHost);
   t_scalar_transpose(m,n,l1,k,E1,E2);
       block=l1*m*n;
	   hipMemcpy(swap3,E2,sizeof(float)*m*n*k*l1,hipMemcpyHostToDevice);
	   idwt_batch<<<block,THREADS>>>(swap3,k2,k1,l1*m*n,swap1);
	   hipMemcpy(A3,swap1,sizeof(float)*m*n*k1*l1,hipMemcpyDeviceToHost);

    for(int i=0;i<m*n*k1*l1;i++){
       			tensor[i].x=A3[i];
       			tensor[i].y=0;
       			}
  //  ifft_batch(m,n,k,l,C,B);
    G=tensor_scalartotensor(m,n,k1,l1,tensor,G);
    printf("test result\n");
   // printctensor(m,n,k,l,E);
   /* for(int i=0;i<m*n*k*l;i++){
    	printf("%f %f \n",E[i].x/(k*l)-tensor[i].x,E[i].y/(k*l)-tensor[i].y);
    }*/
    FILE* fp=fopen("/home/haili/Documents/MATLAB/gpu_based_dwt2/compress_result9_10.txt","w");
    if(!fp){
    	printf("open compress_result.txt error\n");
    	exit(-1);
    }
    for(int i=0;i<m*n*k1*l1;i++){
    	fprintf(fp,"%f\n",G[i].x);
    }
    fclose(fp);
   // printctensor(m,n,k,l,D);
   // fft(m,n,k,l,A,B);
	time2=gettime();
	time3=time2-time1;
	printf("time:%.6f\n",time3);
  //  checkkernel();
	free(A);
	free(E);
	free(B);
	free(D);
	free(C);
	free(S);
	free(U);
	free(V);
	free(S1);
	free(U1);
	free(V1);
	free(tcoo);
	free(row_array);
	free(col_array);
	return 0;
}
