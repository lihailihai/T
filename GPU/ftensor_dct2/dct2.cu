#include "hip/hip_runtime.h"
/*=================dct2=====================================*/
/**
 * Create:22 May,2018
 *
 * @author:haili
 *
 */
#include "dct.cuh"
__global__ void dct2_batch2(float* A,const int m,const int n,float* B){
	__shared__ float mat[THREADS];
	const int tidx=threadIdx.x;
	int tid=tidx;
	int bidx=blockIdx.x;
	const int t_n=blockDim.x;
	double a=0;
	double b=0;
    double temp=0;
  //  int i=0;
   // int j=0;
    while(bidx<BLOCKS){
    for(int i=0;i<m;i++){
    	for(int j=0;j<n;j++){
    tid=tidx;
    temp=0;
    while(tid<m*n){
           int row=tid/n;
           int col=tid%n;
            temp+=A[row*n+col+blockIdx.x*m*n]*cos(pi*(row+0.5)*i/m)*cos(pi*(col+0.5)*j/n);
            tid+=t_n;
    }
    mat[tidx]=temp;
    __syncthreads();
    int k=THREADS/2;
       while(k!=0){
       	if(tidx<k){
       	     mat[tidx]+=mat[tidx+k];
       	     __syncthreads();
       		}
       	    k/=2;
       	}
       a=0;
       b=0;
       if(i==0){
         	a=(double)1/sqrt(double(m));
       }else{
           	a=sqrt((double)2)/sqrt(double(m));
          }
       if(j==0){
               b=(double)1/sqrt(double(n));
       }else{
               b=sqrt((double)2)/sqrt(double(n));
           }
       if(tidx==0){
           	B[i*n+j+blockIdx.x*m*n]=a*b*mat[tidx];
           }
           __syncthreads();

       }
   }
    bidx+=gridDim.x;
    }
}
/*======================idct================================*/
/**
 * @create:22 May,2018
 * @author: haili
 * idct2 shared memmry of size is :THREADS
 * */
__global__ void idct2_batch2(float* A,const int m,const int n,float* B){
	__shared__ float mat[THREADS];
	const int tidx=threadIdx.x;
	int bidx=blockIdx.x;
	const int t_n=blockDim.x;
	const int b_n=gridDim.x;
//	int i=0;
//   int j=0;
	while(bidx<BLOCKS){
	for(int i=0;i<m;i++){
		for(int j=0;j<n;j++){
	int tid=tidx;
	double temp=0;
	double a=0;
	double b=0;
	while(tid<m*n){
		int row=tid/n;
		int col=tid%n;
		if(row==0){
			a=(double)1/sqrt((double)m);
		}else{
			a=sqrt((double)2)/sqrt((double)m);
		}
		if(col==0){
			b=(double)1/sqrt((double)n);
		}else{
			b=sqrt((double)2)/sqrt((double)n);
		}
		temp+=a*b*A[row*n+col+blockIdx.x*m*n]*cos(pi*(i+0.5)*row/m)*cos(pi*(j+0.5)*col/n);
		tid+=t_n;
	}
	mat[tidx]=temp;
	__syncthreads();
	int k=THREADS/2;
	while(k!=0){
		if(tidx<k){
			mat[tidx]+=mat[tidx+k];
			__syncthreads();
		}
		k/=2;
	}

	if(tidx==0){
		B[i*n+j+blockIdx.x*m*n]=mat[tidx];
	}
	__syncthreads();
		}
	}
	bidx+=b_n;
	}
}
