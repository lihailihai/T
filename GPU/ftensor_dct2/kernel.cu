#include "hip/hip_runtime.h"
#include"kernel.h"
/**
 * @odd_even sort
 *  Created on: Apr 16, 2018
 *  len is even
 * @author: haili
 */
__device__ void swap(cdata *a,cdata *b){
   cdata c;
   c=*a;
   *a=*b;
   *b=c;
}

__global__ void odd_even(float* s,cdata* A,int len,int r){
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	cdata a;
        
         for(int i=0;i<len/2;i++){
         if(2*tid+1<len){
         if(A[2*tid].a>A[2*tid+1].a){
            a=A[2*tid]; 
            A[2*tid]=A[2*tid+1];
            A[2*tid+1]=a;
          }
          }
         __syncthreads();
         if(2*tid+2<len){
           if(A[2*tid+1].a>A[2*tid+2].a){
            a=A[2*tid+1]; 
            A[2*tid+1]=A[2*tid+2];
            A[2*tid+2]=a;
           }
          }
          __syncthreads();
         }
         for(int i=0;i<r;i++){
          s[A[i].b]=0;
         }
	}
/**
 * @device tensor_scalar
 * Create on:Apr 17 2018
 * @author: haili
 * the size of tensor is m×n×k×l
 */
__global__ void d_tensor_scalar(hipComplex* A,hipComplex* T,
		const int m,const int n,const int k,const int l){

	int tid=blockDim.x*blockIdx.x+threadIdx.x;
	int t_n=blockDim.x*gridDim.x;
	while(tid<m*n*k*l){
		A[(tid%(m*n))*k*l+tid/(m*n)]=T[tid];
		tid+=t_n;
		__syncthreads();
	}
}
/**
 * @device tensor_scalar to tensor
 * Create on:Apr 17 2018
 * @author: haili
 * the size of tensor is m×n×k×l
 */

__global__ void d_tensor_scalar_tensor(hipComplex* A,hipComplex* T,
		const int m,const int n,const int k,const int l){
	int tid=blockDim.x*blockIdx.x+threadIdx.x;
	int t_n=blockDim.x*gridDim.x;
	while(tid<m*n*k*l){
		A[tid]=T[(tid%(m*n))*k*l+tid/(m*n)];
		tid=tid+t_n;
		__syncthreads();
	}

}
/**
 * @device matview_transopse
 * Create on:Apr 17 2018
 * @author: haili
 * the size of tensor is m×n×k×l
 */
__global__ void d_matview_transpose(hipComplex* A,hipComplex* T,const int m,
		const int n,const int k,const int l){
	int tid=blockDim.x*blockIdx.x+threadIdx.x;
	int t_n=blockDim.x*gridDim.x;
	while(tid<m*n*k*l){
		A[(tid/(m*n))*n*m+(tid%(m*n))/n+((tid%(m*n))%n)*m].x=T[tid].x;
		__syncthreads();
		A[(tid/(m*n))*n*m+(tid%(m*n))/n+(tid%n)*m].y=0-T[tid].y;
		tid=tid+t_n;
		__syncthreads();
	}

}
/**
 * @device csr to matview
 * Create on:Apr 25 2018
 * @author: haili
 * the size of matview is m*k*l × n*k*l
 */
__global__ void d_csrtomatview(hipComplex* A,hipComplex* data,const int* row,
		const int* col, const int m,const int n,const int k,const int l){
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int t_n=blockDim.x*gridDim.x;
	int num=0;

	while(tid<m*k*l){
		if(tid<m*k*l){
		num=row[tid+1]-row[tid];
		__syncthreads();
		for(int i=row[tid];i<row[tid]+num;i++){
			/*A[tid*n+col[i]].x=(float)data[i].x/(k*l);
			A[tid*n+col[i]].y=(float)data[i].y/(k*l);*/
			A[tid*n+col[i]%n]=data[i];
			__syncthreads();

		}
		}
		tid=tid+t_n;
}
}
__global__ void d_zero(hipComplex* A,int len){
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	int t_n=blockDim.x*gridDim.x;
	while(tid<len){
		A[tid].x=0;
		A[tid].y=0;
		tid+=t_n;
	}
	__syncthreads();

}
