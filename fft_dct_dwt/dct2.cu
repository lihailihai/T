#include "hip/hip_runtime.h"
/**
*Create:24 7 2018
*@author:haili
* dim block(x,y)
**/
//----------------------------------------------------------------
#include"dct.h"
__global__ void dct2(const float* A,const int m,const int n,float* B){
        __shared__ float mat[32][32+1];
        const int tid_x=threadIdx.x;
        const int tid_y=threadIdx.y;
        const int t_x=blockDim.x;
        const int t_y=blockDim.y;
        const int bid_x=blockIdx.x;
        float temp=0;
        int tidx=tid_x;
        int tidy=tid_y;	
	for(tidy=tid_y;tidy<n;tidy+=t_y){
        temp=0;
	for(tidx=tid_x;tidx<n*m;tidx+=t_x){
		int row=tidx/n;
                int col=tidx%n;
                temp+=A[row*n+col]*cos(pi*(row+0.5)*bid_x/m)*cos(pi*(col+0.5)*tidy/n); 
	}
	   mat[tid_y][tid_x]=temp;
           __syncthreads();
          int k=t_x/2;
  	  while(k!=0){
		if(tid_x<k){
		mat[tid_y][tid_x]+=mat[tid_y][tid_x+k];	
			}
            __syncthreads();
		k/=2;
	     }
	   float a=0;
		float b=0;
		if(bid_x==0){
			a=(float)1/sqrt(float(m));
			}else{
			a=sqrt((float)2)/sqrt(float(m));
			}
		if(tidy==0){
			b=(float)1/sqrt(float(n));
			}else{
			b=sqrt((float)2)/sqrt(float(n));
			}
	if(tid_x==0){
		B[bid_x*n+tidy]=a*b*mat[tid_y][tid_x];	
		}
	}
}


__global__ void idct2(const float* A,const int m,const int n,float* B){
	 __shared__ float mat[32][32+1];
        const int tid_x=threadIdx.x;
        const int tid_y=threadIdx.y;
        const int t_x=blockDim.x;
        const int t_y=blockDim.y;
        const int bid_x=blockIdx.x;
        float temp=0;
        int tidx=tid_x;
        int tidy=tid_y;
	for(tidy=tid_y;tidy<n;tidy+=t_y){
        temp=0;
	for(tidx=tid_x;tidx<n*m;tidx+=t_x){
		int row=tidx/n;
                int col=tidx%n;
		float a=0;
		float b=0;
		if(row==0){
			a=(float)1/sqrt(float(m));
			}else{
			a=sqrt((float)2)/sqrt(float(m));
			}
		if(col==0){
			b=(float)1/sqrt(float(n));
			}else{
			b=sqrt((float)2)/sqrt(float(n));
			}
                temp+=a*b*A[row*n+col]*cos(pi*(bid_x+0.5)*row/m)*cos(pi*(tidy+0.5)*col/n); 
	}
	   mat[tid_y][tid_x]=temp;
           __syncthreads();
          int k=t_x/2;
  	  while(k!=0){
		if(tid_x<k){
		mat[tid_y][tid_x]+=mat[tid_y][tid_x+k];	
			}
            __syncthreads();
		k/=2;
	     }
	  
	if(tid_x==0){
		B[bid_x*n+tidy]=mat[tid_y][tid_x];	
		}
	}
	
}
