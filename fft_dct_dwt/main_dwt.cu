
#include <hip/hip_runtime.h>
#if 0
#include<cuda.h>
#include<malloc.h>
#include"gettime.c"
#include"tensor.c"
#include"transpose.h"
#include<stdio.h>
#include"dwt.h"

int main(int argc,char* argv[]){

	int k1,l1;
        int batch;
        if(argc==4){
	l1=atoi(argv[1]);
        k1=atoi(argv[2]);
        batch=atoi(argv[3]);}else{
        fprintf(stderr,"[%s:%d] input error,try again",__FUNCTION__,__LINE__);
        }
   
        int k=((k1+7)/2)*2;
	int l=((l1+7)/2)*2;
	int k2=((k1+7)/2);
        int l2=((l1+7)/2);
       
	double time1,time2,time3;
	time1=gettime();
	
	int threads;
	int blocks;
        
//======================create data============================================	
	float* A3=(float*)malloc(sizeof(float)*batch*k1*l1);
	float* A4=(float*)malloc(sizeof(float)*batch*k*l);

	float* A5=(float*)malloc(sizeof(float)*batch*k1*l1);

        srand((unsigned)time(NULL));
         for(int i=0;i<k1*l1*batch;i++){
            A3[i]=(float)rand()/(RAND_MAX/100);
          }
    int block=l1*batch;
    float* swap3;
    float* swap1;
    float* swap4;
    float * E1=(float*)malloc(sizeof(float)*batch*k*l1);
    float * E2=(float*)malloc(sizeof(float)*batch*k*l1);
    float * E3=(float*)malloc(sizeof(float)*batch*k*l1);
    cudaMalloc((void**)&swap3,sizeof(float)*batch*k*l1);
    cudaMalloc((void**)&swap4,sizeof(float)*batch*k*l1);
    cudaMalloc((void**)&swap1,sizeof(float)*batch*k1*l1);
    time1=gettime();
//=========================dwt2================================================
    cudaMemcpy(swap1,A3,sizeof(float)*batch*k1*l1,cudaMemcpyHostToDevice);
    dwt_batch<<<block,THREADS>>>(swap1,k1,l1*batch,swap3);
   // cudaMemcpy(E1,swap3,sizeof(float)*batch*k*l1,cudaMemcpyDeviceToHost);
   // t_scalar_transpose(k,l1,batch,E1,E2);
    if(batch*k*l1<512){
        	threads=k*l1*batch;
        	blocks=1;
        }else{
        	threads=512;
        	blocks=(batch*k*l1%512==0)?batch*k*l1/512:batch*k*l1/512+1;
        }
    d_batch_transpose<<<blocks,threads>>>(swap4,swap3,l1,k,batch);
    //cudaMemcpy(E3,swap4,sizeof(float)*batch*k*l1,cudaMemcpyDeviceToHost); 
    block=k*batch;
    float* swap6;
    float* swap7;
    cudaMalloc((void**)&swap6,sizeof(float)*batch*k*l);
    cudaMalloc((void**)&swap7,sizeof(float)*batch*k*l);
   // cudaMemcpy(swap3,E2,sizeof(float)*batch*k*l1,cudaMemcpyHostToDevice);
    dwt_batch<<<block,THREADS>>>(swap4,l1,k*batch,swap6);
   // cudaMemcpy(A4,swap6,sizeof(float)*batch*k*l,cudaMemcpyDeviceToHost);
    if(batch*k*l<512){
        	threads=k*l*batch;
        	blocks=1;
        }else{
        	threads=512;
        	blocks=(batch*k*l%512==0)?batch*k*l/512:batch*k*l/512+1;
        }
    //d_batch_transpose<<<blocks,threads>>>(swap7,swap6,k,l,batch);
    cudaDeviceSynchronize();
    time2=gettime();
//============================================================================
//======================idwt2=================================================
           block=k*batch;
          // d_batch_transpose<<<blocks,threads>>>(swap6,swap7,l,k,batch);
	  // cudaMemcpy(swap6,A4,sizeof(float)*batch*k*l,cudaMemcpyHostToDevice);
	   idwt_batch<<<block,THREADS>>>(swap6,l2,l1,k*batch,swap3);
	  // cudaMemcpy(E1,swap3,sizeof(float)*batch*k*l1,cudaMemcpyDeviceToHost);
          // t_scalar_transpose(l1,k,batch,E1,E2);
            if(batch*k*l1<512){
        	threads=k*l1*batch;
        	blocks=1;
            }else{
        	threads=512;
        	blocks=(batch*k*l1%512==0)?batch*k*l1/512:batch*k*l1/512+1;
               }
            d_batch_transpose<<<blocks,threads>>>(swap4,swap3,k,l1,batch);
            block=l1*batch;
	   //cudaMemcpy(swap3,E2,sizeof(float)*batch*k*l1,cudaMemcpyHostToDevice);
	   idwt_batch<<<block,THREADS>>>(swap4,k2,k1,l1*batch,swap1);
	   cudaMemcpy(A5,swap1,sizeof(float)*batch*k1*l1,cudaMemcpyDeviceToHost);
           time3=gettime();
//===============================================================================
       /* for(int i=0;i<batch*k1*l1;i++){	
       printf("%f\n",A3[i]-A5[i]);
        }*/
    printf("%d %d %d %f %f\n",k1,l1,batch,time2-time1,time3-time2);
    cudaFree(swap3);
    cudaFree(swap1);
    cudaFree(swap6);
    cudaFree(swap7);
    free(A3);
    free(A4); 
    free(A5);
    free(E1);
    free(E2);
	return 0;
}
#endif
