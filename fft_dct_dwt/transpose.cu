
#include <hip/hip_runtime.h>
/**
 * @device matview_transopse
 * Create on:Apr 17 2018
 * @author: haili
 * the size of tensor is m×n×k×l
 */
__global__ void d_batch_transpose(float* A,float* T,const int m,
		const int n,const int batch){
	int tid=blockDim.x*blockIdx.x+threadIdx.x;
	int t_n=blockDim.x*gridDim.x;
	while(tid<m*n*batch){
		A[(tid/(m*n))*n*m+(tid%(m*n))/n+((tid%(m*n))%n)*m]=T[tid];
		tid=tid+t_n;
		__syncthreads();
	}
   

}
