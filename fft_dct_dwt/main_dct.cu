
#include <hip/hip_runtime.h>
//=====================================================================
//  DCT2 BATCH 
//   AUTHOR:HAILI
//======================================================================
#if 0
#include"dct.h"
#include"gettime.c"
#include<cuda_runtime.h>
int main(int argc,char *argv[])
{
    int m;
    int n;
    int batch;
    if(argc==4){
         m=atoi(argv[1]);
         n=atoi(argv[2]);
         batch=atoi(argv[3]);
       }else{
          fprintf(stderr,"[%s:%d]input error,try again!\n",__FUNCTION__,__LINE__);   
           exit(-1);
              }
       float* d_data;
       float* d_result;
       cudaMalloc((void**)&d_data,sizeof(float)*m*n*batch);
       cudaMalloc((void**)&d_result,sizeof(float)*m*n*batch);
       float* data=(float*)malloc(sizeof(float)*n*m*batch);
       float* result=(float*)malloc(sizeof(float)*m*n*batch*2);
       float* data_test=(float*)malloc(sizeof(float)*n*m*batch);
//---------------create data----------------------------------------------
    srand((unsigned(time(NULL))));
    for(int i=0;i<m*n*batch;i++){
          data[i]=i;  
    //data[i]=(float)rand()/(RAND_MAX/100); 
      }
   /* for(int i=0;i<m*n*batch;i++){
       printf("%f\n",data[i]);
      }*/
//--------------------dct2-----------------------------------------------
    double time1,time2,time3,time4;
    cudaMemcpy(d_data,data,sizeof(float)*m*n*batch,cudaMemcpyHostToDevice);
    int blocks=batch;
    int threads=THREADS;
    time1=gettime();
    dct2_batch<<<blocks,threads>>>(d_data,m,n,d_result,batch);
    
   // printf("dsajdslk\n");
    cudaMemcpy(result,d_result,sizeof(float)*batch*m*n,cudaMemcpyDeviceToHost);
    time4=gettime();
    /* for(int i=0;i<m*n*batch;i++){
       printf("%f\n",result[i]);
       }*/

    time2=gettime();
    idct2_batch<<<blocks,threads>>>(d_result,m,n,d_data,batch);
    
    //printf("dhahah\n");
    cudaMemcpy(data_test,d_data,sizeof(float)*batch*m*n,cudaMemcpyDeviceToHost);
    time3=gettime();
     /*for(int i=0;i<m*n*batch;i++){
       printf("%f   %f\n",data_test[i]-data[i],data_test[i]);
         }*/
    printf("%d %d %d %f %f\n",m,n,batch,time4-time1,time3-time2);
}
#endif
