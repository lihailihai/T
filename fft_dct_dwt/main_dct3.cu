#include"dct.h"
int main(int argc,char **argv){
	int m;
	int n;
	if(argc==3){
		m=atoi(argv[1]);	
		n=atoi(argv[2]);	
		}else{
		fprintf(stderr,"[%s:%d]input format error,try again!",__FUNCTION__,__LINE__);
		}
//*******************create data************************************
	float* data=(float*)malloc(sizeof(float)*n*m);
	float* out=(float*)malloc(sizeof(float)*m*n);
	for(int i=0;i<m*n;i++){
		data[i]=i;
	}
//******************memery malloc***********************************
	float* d_data=NULL;
	float* d_out=NULL;
	hipMalloc((void**)&d_data,sizeof(float)*m*n);
	hipMalloc((void**)&d_out,sizeof(float)*m*n);
//******************dct2*******************************************
	hipMemcpy(d_data,data,sizeof(float)*m*n,hipMemcpyHostToDevice);
        dim3 block(32,32,1);
        dim3 grid(m,1,1);
       
	dct2<<<grid,block>>>(d_data,m,n,d_out);
	hipMemcpy(out,d_out,sizeof(float)*m*n,hipMemcpyDeviceToHost);
	for(int j=0;j<m;j++){
	for(int i=0;i<n;i++){
	printf("%f ",out[j*n+i]);
	}
	printf("\n");
	}
	printf("------------------------------\n");
//****************idct2*******************************************
	hipMemcpy(d_out,out,sizeof(float)*m*n,hipMemcpyHostToDevice);
	idct2<<<grid,block>>>(d_out,m,n,d_data);
	hipMemcpy(out,d_data,sizeof(float)*m*n,hipMemcpyDeviceToHost);
	for(int j=0;j<m;j++){
	for(int i=0;i<n;i++){
	printf("%f ",out[j*n+i]);
	}
	printf("\n");
	}
}
