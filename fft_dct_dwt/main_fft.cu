
#include <hip/hip_runtime.h>
/***************************************************
*   test fft 
*   AUTHOR:haili
*
****************************************************/
#if 0
#include<cuda.h>
#include<stdlib.h>
#include<stdio.h>
#include"fft.h"
#include"gettime.c"
#include<time.h>
int main(int argc,char* argv[]){
      int m,n,batch;
      if(argc==4){
        m=atoi(argv[1]);
        n=atoi(argv[2]);
        batch=atoi(argv[3]);
       }else{
           fprintf(stderr,"[%s:%d]input err!try again!\n",__FUNCTION__,__LINE__);
           exit(-1);
           }
 //----------create data------------------------------
      srand((unsigned)time(NULL));
      cuComplex* data=(cuComplex*)malloc(sizeof(cuComplex)*batch*m*n);
      for(int i=0;i<m*n*batch;i++){
            data[i].x=(float)rand()/(RAND_MAX/100);
            data[i].y=(float)rand()/(RAND_MAX/100);
          }
//----------fft2-------------------------------------
//      for(int i=0;i<m*n*batch;i++){
//          printf("%f,%f\n",data[i].x,data[i].y);
//          }
    double time1,time2,time3,time4;
    cuComplex* result=(cuComplex*)malloc(sizeof(cuComplex)*batch*n*m);
    time1=gettime();
    fft2_batch(m,n,data,result,batch);
    time2=gettime();
//      for(int i=0;i<m*n*batch;i++){
//          printf("%f,%f\n",result[i].x,result[i].y);
//          }
    time4=gettime();
    ifft2_batch(m,n,result,data,batch);
    time3=gettime();
    printf("%d %d %d %f %f\n",m,n,batch,time2-time1,time3-time4);
//      for(int i=0;i<m*n*batch;i++){
//          printf("%f,%f\n",data[i].x/6,data[i].y/6);
//          }
return 0;
}
#endif
