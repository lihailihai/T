
#include <hip/hip_runtime.h>
#if 0
#include<cuda.h>
#include<malloc.h>
#include"gettime.c"
#include"tensor.c"
#include"transpose.h"
#include<stdio.h>
#include"dct.h"

int main(int argc,char* argv[]){

	int k1,l1;
        int batch;
        if(argc==4){
	k1=atoi(argv[1]);
        l1=atoi(argv[2]);
        batch=atoi(argv[3]);}else{
        fprintf(stderr,"[%s:%d] input error,try again",__FUNCTION__,__LINE__);
        }
   
	double time1,time2,time3;
	time1=gettime();
	
	int threads;
	int blocks;
        
//======================create data============================================	
	float* A3=(float*)malloc(sizeof(float)*batch*k1*l1);
	float* A4=(float*)malloc(sizeof(float)*batch*k1*l1);
	float* A5=(float*)malloc(sizeof(float)*batch*k1*l1*2);
        srand((unsigned)time(NULL));
         for(int i=0;i<k1*l1*batch;i++){
            A3[i]=(float)rand()/(RAND_MAX/100);
          }
    int block=k1*batch;
    const int BLOCK=30000;
    float* swap3;
    float* swap1;
    float* swap4;
    float * E1=(float*)malloc(sizeof(float)*batch*k1*l1);
    float * E2=(float*)malloc(sizeof(float)*batch*k1*l1);
    float * E3=(float*)malloc(sizeof(float)*batch*k1*l1);
    cudaMalloc((void**)&swap1,sizeof(float)*batch*k1*l1);
    cudaMalloc((void**)&swap3,sizeof(float)*batch*k1*l1);
    cudaMalloc((void**)&swap4,sizeof(float)*batch*k1*l1);
    float* swap6;
    float* swap7;
    cudaMalloc((void**)&swap6,sizeof(float)*batch*k1*l1);
    cudaMalloc((void**)&swap7,sizeof(float)*batch*k1*l1);
    time1=gettime();
//=========================dct2================================================
    cudaMemcpy(swap1,A3,sizeof(float)*batch*k1*l1,cudaMemcpyHostToDevice);
    dct_batch<<<BLOCK,THREADS>>>(swap1,l1,k1*batch,swap3);
   // cudaMemcpy(E1,swap3,sizeof(float)*batch*k1*l1,cudaMemcpyDeviceToHost);
   // t_scalar_transpose(l1,k1,batch,E1,E2);
    if(batch*k1*l1<512){
        	threads=k1*l1*batch;
        	blocks=1;
        }else{
        	threads=512;
        	blocks=(batch*k1*l1%512==0)?batch*k1*l1/512:batch*k1*l1/512+1;
        }
    d_batch_transpose<<<blocks,threads>>>(swap4,swap3,k1,l1,batch);
    //cudaMemcpy(E3,swap4,sizeof(float)*batch*k1*l1,cudaMemcpyDeviceToHost); 
    block=l1*batch;
   
   // cudaMemcpy(swap3,E2,sizeof(float)*batch*k1*l1,cudaMemcpyHostToDevice);
    dct_batch<<<BLOCK,THREADS>>>(swap4,k1,l1*batch,swap6);
   // cudaMemcpy(A4,swap6,sizeof(float)*batch*k1*l1,cudaMemcpyDeviceToHost);
    d_batch_transpose<<<blocks,threads>>>(swap7,swap6,l1,k1,batch);
    cudaDeviceSynchronize();
    time2=gettime();
//============================================================================
//======================idct2=================================================
           block=l1*batch;
           d_batch_transpose<<<blocks,threads>>>(swap6,swap7,k1,l1,batch);
	  // cudaMemcpy(swap6,A4,sizeof(float)*batch*k1*l1,cudaMemcpyHostToDevice);
	   idct_batch<<<BLOCK,THREADS>>>(swap6,k1,l1*batch,swap3);
	   //cudaMemcpy(E1,swap3,sizeof(float)*batch*k1*l1,cudaMemcpyDeviceToHost);
           //t_scalar_transpose(l1,k1,batch,E1,E2);
           d_batch_transpose<<<blocks,threads>>>(swap4,swap3,l1,k1,batch);
            block=k1*batch;
	  // cudaMemcpy(swap3,E2,sizeof(float)*batch*k1*l1,cudaMemcpyHostToDevice);
	   idct_batch<<<BLOCK,THREADS>>>(swap4,l1,k1*batch,swap1);
	   cudaMemcpy(A5,swap1,sizeof(float)*batch*k1*l1,cudaMemcpyDeviceToHost);
           time3=gettime();
//===============================================================================
      /*  for(int i=0;i<batch*k1*l1;i++){	
       printf("%f\n",A3[i]-A5[i]);
        }*/
    printf("%d %d %d %f %f\n",k1,l1,batch,time2-time1,time3-time2);
     

  /*  cudaFree(swap3);
    cudaFree(swap1);
    cudaFree(swap4);
    cudaFree(swap6);
    cudaFree(swap7);
    free(A3);
    free(A4); 
    free(A5);
    free(E1);
    free(E2);*/
	return 0;
}
#endif
