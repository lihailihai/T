#include "hip/hip_runtime.h"
/**==============================dwt_batch====================================
 * create on: 2,Jun,2018
 * @author: haili
 *============================================================================
 */
#include "dwt.h"
__device__ void exch(int tid,int l,float* A,float* B){
	if(tid<l){
					B[tid+l]=A[tid];
					B[tid]=A[l-tid-1];
					B[tid+2*l]=A[l-tid-1];
					__syncthreads();
				}
}
__global__ void dwt_batch(float* A,const int len,const int batch,float* B){
	const int tidx=threadIdx.x;
	const int tid_n=blockDim.x;
	int tid=tidx;
	const int filterlen=8;
	const int declen=(len+filterlen-1)/2;
	__shared__ float lowfilter[filterlen];
	//lowfilter[]={0.2304,0.7148,0.6309,-0.0280,-0.1870,0.0308,0.0329,-0.0106};
	lowfilter[7]=0.2303778133088964;lowfilter[6]=0.7148465705529154;
	lowfilter[5]=0.6308807679398597;lowfilter[4]=-0.0279837694168599;
	lowfilter[3]=-0.1870348117190931;lowfilter[2]=0.0308413818355607;
	lowfilter[1]=0.0328830116668852;lowfilter[0]=-0.0105974017850890;
	__shared__ float highfilter[filterlen];
   // highfilter[]={-0.0106,-0.0329,0.0308,0.1870,-0.0280,-0.6309,0.7148,-0.2304};
	highfilter[7]=-0.0105974017850890;highfilter[6]=-0.0328830116668852;
	highfilter[5]=0.0308413818355607;highfilter[4]=0.1870348117190931;
	highfilter[3]=-0.0279837694168599;highfilter[2]=-0.6308807679398597;
	highfilter[1]=0.7148465705529154;highfilter[0]=-0.2303778133088964;
	__shared__ float temp[THREADS];
	__shared__ float temp1[THREADS];
//	int i=0;
	switch(len)
	{
	case 1:{

		for(int i=0;i<4;i++){
		temp[tidx]=A[blockIdx.x];
	    float tmp=0;
		float tmp1=0;
		tid=tidx;
		while(tid<8){
		tmp+=temp[tid+i]*lowfilter[7-tid];
		tmp1+=temp[tid+i]*highfilter[7-tid];
		tid+=tid_n;
		}
                 __syncthreads();
		temp[tidx]=tmp;
		temp1[tidx]=tmp1;
		__syncthreads();
		int k=THREADS/2;
		while(k!=0){
			if(tidx<k){
				temp[tidx]+=temp[tidx+k];
				temp1[tidx]+=temp1[tidx+k];
			}
			__syncthreads();
			k/=2;
		  }
		if(tidx==0){
			B[i+blockIdx.x*8]=temp[tidx];
			B[i+4+blockIdx.x*8]=temp1[tidx];
		}
		__syncthreads();
		}
		break;
	   }
	case 2 :{
		for(int i=0;i<4;i++){
		float tmp=0;
		float tmp1=0;
		tid=tidx;
		if(tid<len){
			temp[tid+len]=A[tid+blockIdx.x*len];
			temp[tid]=A[len-tid-1+blockIdx.x*len];
			temp[tid+2*len]=A[len-tid-1+blockIdx.x*len];
				}
		__syncthreads();
		if(tid<len*3){
			temp1[tid+len*3]=temp[tid];
			temp1[tid]=temp[3*len-tid-1];
			temp1[tid+6*len]=temp[3*len-tid-1];
		}
		__syncthreads();
		while(tid<8){
		tmp+=temp1[tid+2+i*2]*lowfilter[7-tid];
		tmp1+=temp1[tid+2+i*2]*highfilter[7-tid];
		tid+=tid_n;
		}

		temp[tidx]=tmp;
		temp1[tidx]=tmp1;
		__syncthreads();
		int k=THREADS/2;
		while(k!=0){
		if(tidx<k){
		  temp[tidx]+=temp[tidx+k];
		  temp1[tidx]+=temp1[tidx+k];
					}
		 __syncthreads();
			k/=2;
			  }
		/*tmp=temp[0];
		printf("dshd%f\n",tmp);*/
		if(tidx==0){
		  B[i+blockIdx.x*8]=temp[tidx];
		  B[i+4+blockIdx.x*8]=temp1[tidx];
		 }
		__syncthreads();
		}
		break;
	}
	case 3 :{
		for(int i=0;i<5;i++){
			float tmp=0;
			float tmp1=0;
			tid=tidx;
			if(tid<len){
				temp[tid+len]=A[tid+blockIdx.x*len];
				temp[tid]=A[len-tid-1+blockIdx.x*len];
				temp[tid+2*len]=A[len-tid-1+blockIdx.x*len];
						}
			__syncthreads();
			if(tid<len*3){
				temp1[tid+len*3]=temp[tid];
				temp1[tid]=temp[3*len-tid-1];
				temp1[tid+6*len]=temp[3*len-tid-1];
			}
			__syncthreads();
			while(tid<8){
			tmp+=temp1[tid+6+i*2]*lowfilter[7-tid];
			tmp1+=temp1[tid+6+i*2]*highfilter[7-tid];
			tid+=tid_n;
			}
			temp[tidx]=tmp;
			temp1[tidx]=tmp1;
			__syncthreads();
			int k=THREADS/2;
			while(k!=0){
			if(tidx<k){
			temp[tidx]+=temp[tidx+k];
			temp1[tidx]+=temp1[tidx+k];
							}
			 __syncthreads();
			k/=2;
			}
			if(tidx==0){
   		  B[i+blockIdx.x*10]=temp[tidx];
   		  B[i+5+blockIdx.x*10]=temp1[tidx];
				 }
		__syncthreads();
				}
		break;
	}
	case 4 :{
		for(int i=0;i<5;i++){
			float tmp=0;
			float tmp1=0;
			tid=tidx;
			if(tid<len){
			temp[tid+len]=A[tid+blockIdx.x*len];
			temp[tid]=A[len-tid-1+blockIdx.x*len];
		    temp[tid+2*len]=A[len-tid-1+blockIdx.x*len];
						}
			__syncthreads();
			if(tid<len*3){
			temp1[tid+len*3]=temp[tid];
			temp1[tid]=temp[3*len-tid-1];
			temp1[tid+6*len]=temp[3*len-tid-1];
			}
			__syncthreads();
			while(tid<8){
			tmp+=temp1[tid+10+i*2]*lowfilter[7-tid];
			tmp1+=temp1[tid+10+i*2]*highfilter[7-tid];
			tid+=tid_n;
			}
			 temp[tidx]=tmp;
			 temp1[tidx]=tmp1;
			__syncthreads();
			int k=THREADS/2;
			while(k!=0){
				if(tidx<k){
				temp[tidx]+=temp[tidx+k];
				temp1[tidx]+=temp1[tidx+k];
							}
				__syncthreads();
				k/=2;
				}
			__syncthreads();
			tmp1=temp1[0];
	      if(tidx==0){
		   	B[i+blockIdx.x*10]=temp[tidx];
	    	 B[i+5+blockIdx.x*10]=temp1[tidx];
						 }
		  __syncthreads();
				}
		break;
	}
	case 5 :{
		for(int i=0;i<6;i++){
			float tmp=0;
			float tmp1=0;
			tid=tidx;
			temp[tidx]=0;
			temp1[tidx]=0;
			if(tid<len){
			temp[tid+len]=A[tid+blockIdx.x*len];
			temp[tid]=A[len-tid-1+blockIdx.x*len];
		    temp[tid+2*len]=A[len-tid-1+blockIdx.x*len];
							}
			__syncthreads();
		 if(tid<len*3){
			temp1[tid+len*3]=temp[tid];
			temp1[tid]=temp[3*len-tid-1];
			temp1[tid+6*len]=temp[3*len-tid-1];
					}
			__syncthreads();
		while(tid<8){
			tmp+=temp1[tid+14+i*2]*lowfilter[7-tid];
			tmp1+=temp1[tid+14+i*2]*highfilter[7-tid];
			tid+=tid_n;
					}
		    temp[tidx]=0;
		    temp1[tidx]=0;
		    __syncthreads();
			temp[tidx]=tmp;
			temp1[tidx]=tmp1;
			__syncthreads();
	    int k=THREADS/2;
	    while(k!=0){
			if(tidx<k){
			   temp[tidx]+=temp[tidx+k];
			   temp1[tidx]+=temp1[tidx+k];
									}
			 __syncthreads();
					k/=2;
						}
		 if(tidx==0){
			 	B[i+blockIdx.x*12]=temp[tidx];
			    B[i+6+blockIdx.x*12]=temp1[tidx];
								 }
		 __syncthreads();
						}
		break;
	}
	default:{
	for(int i=0;i<declen;i++){
		float tmp=0;
		float tmp1=0;
		tid=tidx;
		while(tid<filterlen){
		int p=2*i-tid+1;
		if((p<0) && (p>=(-filterlen+1))){
			tmp+=A[-p-1+blockIdx.x*len]*lowfilter[tid];
			tmp1+=A[-p-1+blockIdx.x*len]*highfilter[tid];
		}else {
			if((p>len-1) && (p<=len+filterlen-2)){
			tmp+=A[2*len-1-p+blockIdx.x*len]*lowfilter[tid];
			tmp1+=A[2*len-1-p+blockIdx.x*len]*highfilter[tid];
		}else{
			if((p>=0) && (p<=len-1)){
			tmp+=A[p+blockIdx.x*len]*lowfilter[tid];
			tmp1+=A[p+blockIdx.x*len]*highfilter[tid];
		}else{
			tmp+=0;
		    tmp1+=0;
		}
	         }
		           }
		//printf("%f\n",tmp);
		tid+=tid_n;
		}
		temp[tidx]=tmp;
		temp1[tidx]=tmp1;
		__syncthreads();
		int k=THREADS/2;
		while(k!=0){
			if(tidx<k){
				temp[tidx]+=temp[tidx+k];
				temp1[tidx]+=temp1[tidx+k];	
			}
                   __syncthreads();
			k/=2;
		}
		
	//	tmp=temp[0];
	 //   printf("%f\n",tmp);
		if(tidx==0){
		B[i+blockIdx.x*declen*2]=temp[tidx];
		B[i+declen+blockIdx.x*declen*2]=temp1[tidx];
		}
           __syncthreads();
	}
	break;
	}
	}
}
/**
 * ================================idwt1_batch===============================
 * @A.length=len * batch
 * @B.length=((len/2==0)?2*len-filterlen+2:2*len-filterlen+1)*batch
 * @len=declen_L or  declen_H
 * ==========================================================================
 */
__global__ void idwt_batch(float* A,const int len,const int reclen,const int batch,float* B){
	const int tidx=threadIdx.x;
	const int tid_n=blockDim.x;
	int tid=tidx;
	const int filterlen=8;
	__shared__ float lowfilter[filterlen];
	//lowfilter[]={0.2304,0.7148,0.6309,-0.0280,-0.1870,0.0308,0.0329,-0.0106};
	lowfilter[0]=0.2303778133088964;lowfilter[1]=0.7148465705529154;
	lowfilter[2]=0.6308807679398597;lowfilter[3]=-0.0279837694168599;
	lowfilter[4]=-0.1870348117190931;lowfilter[5]=0.0308413818355607;
	lowfilter[6]=0.0328830116668852;lowfilter[7]=-0.0105974017850890;

	__shared__ float highfilter[filterlen];
	 // highfilter[]={-0.0106,-0.0329,0.0308,0.1870,-0.0280,-0.6309,0.7148,-0.2304};
	highfilter[0]=-0.0105974017850890;highfilter[1]=-0.0328830116668852;
	highfilter[2]=0.0308413818355607;highfilter[3]=0.1870348117190931;
	highfilter[4]=-0.0279837694168599;highfilter[5]=-0.6308807679398597;
	highfilter[6]=0.7148465705529154;highfilter[7]=-0.2303778133088964;
	__shared__ float temp[THREADS];
//	const int reclen=2*len-filterlen+2;
	for(int i=0;i<reclen;i++){
		tid=tidx;
		float tmp=0;
		while(tid<len){
			int p=i-2*tid+filterlen-2;
			if((p>=0) && (p<filterlen)){
			 tmp+=lowfilter[p]*A[tid+blockIdx.x*2*len]+highfilter[p]*A[tid+len+blockIdx.x*2*len];
			}
			tid+=tid_n;
		}
		temp[tidx]=tmp;
		__syncthreads();
		int k=THREADS/2;
		while(k!=0){
			if(tidx<k){
				temp[tidx]+=temp[tidx+k];	
			}
                  __syncthreads();
			k/=2;
		}
		if(tidx==0){
			B[i+blockIdx.x*reclen]=temp[tidx];
		}
		__syncthreads();
	}
}
