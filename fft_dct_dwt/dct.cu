#include "hip/hip_runtime.h"
#include "dct.h"
/**
 * implement  a set of dct with the same size.
 * create:19 May 2018
 * @author: haili
 *threadnum must be even ,threadnum < len
 */
 __global__ void dct_batch(float* A,const int len,const int batch,float* B){
         __shared__  float tmp[THREADS];
        const int tidx=threadIdx.x;
	const int t_n=blockDim.x;
        const int bidx=blockIdx.x;
        const int b_n=gridDim.x;
        int Bid=bidx;
	double temp=0.0;
	double a=0.0;
        while(Bid<batch){
	for(int i=0;i<len;i++){
		int Tid2=tidx;
		temp=0.0;
		a=0;
		if(i==0){
			a=(double)1/sqrt((double)len);
		}else{
			a=sqrt((double)2)/sqrt((double)len);
		}
		while(Tid2<len){
		//temp+=A[Tid2+blockIdx.x*len]*cos(pi*(Tid2+0.5)*i/len);
                temp+=A[Tid2+Bid*len]*cos(pi*(Tid2+0.5)*i/len);
		Tid2+=t_n;
		}
		tmp[tidx]=temp;
		__syncthreads();
	    int k=THREADS/2;
		while(k!=0){
			if(tidx<k){
				tmp[tidx]+=tmp[tidx+k];
			}
			__syncthreads();
			k/=2;
		  }
		if(tidx==0){
		//	B[i+blockIdx.x*len]=a*tmp[tidx];
                      B[i+Bid*len]=a*tmp[tidx];
			}
}            Bid+=b_n;
	}

}
 /*======================idct================================*/
 /**
  * @create:22 May,2018
  * @author: haili
  * idct2 shared memmry of size is :THREADS
  * */
 __global__ void idct_batch(float* A,const int len,const int batch,float* B){
 	__shared__ float mat[THREADS];
 	const int tidx=threadIdx.x;
 	int t_n=blockDim.x;
        const int bidx=blockIdx.x;
        const int b_n=gridDim.x;
        int Bid=bidx;
        while(Bid<batch){
 	for(int j=0;j<len;j++){
 	int tid=tidx;
 	double temp=0;
 	double a=0;
 	while(tid<len){
 		if(tid==0){
 			a=(double)1/sqrt((double)len);
 		}else{
 			a=sqrt((double)2)/sqrt((double)len);
 		}
 	//	temp+=a*A[tid+blockIdx.x*len]*cos((pi*(j+0.5)*tid)/len);
               temp+=a*A[tid+Bid*len]*cos((pi*(j+0.5)*tid)/len);
 		tid+=t_n;
 	}
 	mat[tidx]=temp;
 	__syncthreads();
 	int k=THREADS/2;
 	while(k!=0){
 		if(tidx<k){
 			mat[tidx]+=mat[tidx+k];
 		}
 		__syncthreads();
 		k/=2;
 	}

 	if(tidx==0){
 	//	B[j+blockIdx.x*len]=mat[tidx];
            B[j+Bid*len]=mat[tidx];
 	}
 		}
         Bid+=b_n;
        }
 	}


